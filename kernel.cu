#include "hip/hip_runtime.h"
﻿/*
 *  file name: kernel.cu
 *
 *  kernel.cu contains the code that realize some common used matrix operations in CUDA
 *
 *  this is a toy program for learning CUDA, some functions are reusable in other project
 *
 */
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <math.h>
#define BLOCK_SIZE 32

struct statistical_para {
    double std;
    double mean;
    double confi_inter_left;
    double confi_inter_right;
};

statistical_para count_statistic(int* matrix, int size) {
    statistical_para ret{};
    int sum = 0;
    double sum_sq = 0;

    // Calculate sum of elements
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            sum += matrix[i * size + j];
        }
    }
    ret.mean = static_cast<double>(sum) / (size * size);

    // Calculate sum of squared differences from mean
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            sum_sq += (static_cast<double>(matrix[i * size + j]) - ret.mean) *
                (static_cast<double>(matrix[i * size + j]) - ret.mean);
        }
    }
    ret.std = sqrt(sum_sq);

    // Calculate confidence interval
    ret.confi_inter_left = ret.mean - 1.96 * (ret.std / sqrt(size));
    ret.confi_inter_right = ret.mean + 1.96 * (ret.std / sqrt(size));

    return ret;
}

__global__ void gpu_matrix_mult(int* a, int* b, int* c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (col < k && row < m)
    {
        for (int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
}

__global__ void gpu_square_matrix_mult(int* d_a, int* d_b, int* d_result, int n)
{
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int tmp = 0;
    int idx;

    for (int sub = 0; sub < gridDim.x; ++sub)
    {
        idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        if (idx >= n * n)
        {
            // n may not divisible by BLOCK_SIZE
            tile_a[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_a[threadIdx.y][threadIdx.x] = d_a[idx];
        }

        idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
        if (idx >= n * n)
        {
            tile_b[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_b[threadIdx.y][threadIdx.x] = d_b[idx];
        }
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    if (row < n && col < n)
    {
        d_result[row * n + col] = tmp;
    }
}

__global__ void gpu_matrix_transpose(int* mat_in, int* mat_out, unsigned int rows, unsigned int cols)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < cols && idy < rows)
    {
        unsigned int pos = idy * cols + idx;
        unsigned int trans_pos = idx * rows + idy;
        mat_out[trans_pos] = mat_in[pos];
    }
}


void cpu_matrix_mult(int* h_a, int* h_b, int* h_result, int m, int n, int k) {
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            int tmp = 0.0;
            for (int h = 0; h < n; ++h)
            {
                tmp += h_a[i * n + h] * h_b[h * k + j];
            }
            h_result[i * k + j] = tmp;
        }
    }
}

void write_matrix_to_file(int* matrix, int size, const char* filename) {
    FILE* file = fopen(filename, "w");
    if (file == NULL) {
        printf("Error opening file.\n");
        return;
    }

    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            fprintf(file, "%d ", matrix[i * size + j]);
        }
    }

    fclose(file);
}

void algorith_main(int n) {
    srand(3333);

    // allocate memory in host RAM, h_cc is used to store CPU result
    int* h_a, * h_b, * h_c, * h_cc;
    hipHostMalloc((void**)&h_a, sizeof(int) * n * n);
    hipHostMalloc((void**)&h_b, sizeof(int) * n * n);
    hipHostMalloc((void**)&h_c, sizeof(int) * n * n);
    hipHostMalloc((void**)&h_cc, sizeof(int) * n * n);

    // random initialize matrix A
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            h_a[i * n + j] = rand() % 1024;
        }
    }
    write_matrix_to_file(h_a, n, "matrixA.txt");

    // random initialize matrix B
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            h_b[i * n + j] = rand() % 1024;
        }
    }
    write_matrix_to_file(h_b, n, "matrixB.txt");

    float gpu_elapsed_time_ms, cpu_elapsed_time_ms;

    printf("Block size is: %d\n", BLOCK_SIZE);

    for (int size = 100; size <= n; size += 100)
    {
        // some events to count the execution time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // start to count execution time of GPU version

        hipEventRecord(start, 0);
        clock_t start_gpu = clock();

        // Allocate memory space on the device 
        int* d_a, * d_b, * d_c;
        hipMalloc((void**)&d_a, sizeof(int) * size * size);
        hipMalloc((void**)&d_b, sizeof(int) * size * size);
        hipMalloc((void**)&d_c, sizeof(int) * size * size);

        // copy matrix A and B from host to device memory
        hipMemcpy(d_a, h_a, sizeof(int) * size * size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b, sizeof(int) * size * size, hipMemcpyHostToDevice);

        unsigned int grid_rows = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
        unsigned int grid_cols = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
        dim3 dimGrid(grid_cols, grid_rows);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

        // Launch kernel 


        gpu_square_matrix_mult << <dimGrid, dimBlock >> > (d_a, d_b, d_c, size);

        // Transefr results from device to host 
        hipMemcpy(h_c, d_c, sizeof(int) * size * size, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        // time counting terminate

        // compute time elapse on GPU computing

        gpu_elapsed_time_ms = clock() - start_gpu;

        printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on GPU: %f s\n", size, size, size, size, (gpu_elapsed_time_ms / CLOCKS_PER_SEC) * pow(10.0, 6) / 1000 / 1000);



        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
        // printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on GPU: %f s - using CudaEvent\n\n", n, n, n, n, gpu_elapsed_time_ms / 1000);


        clock_t start_cpu = clock();

        cpu_matrix_mult(h_a, h_b, h_cc, size, size, size);

        cpu_elapsed_time_ms = clock() - start_cpu;

        printf("Time elapsed on matrix multiplication on CPU: %f s.\n\n", (cpu_elapsed_time_ms / CLOCKS_PER_SEC) * pow(10.0, 6) / 1000 / 1000);
        
        if (size == n)
        {
            write_matrix_to_file(h_c, n, "matrix_result.txt");

            statistical_para st;
            st = count_statistic(h_a, size);

            printf("\nStatistical parameters for matrix A\n");
            printf("Mean of matrix: %.2f\n", st.mean);
            printf("Standard deviation of matrix: %.2f\n", st.std);
            printf("Confidence interval with 95%% confidence: (%.2f, %.2f)\n", st.confi_inter_left, st.confi_inter_right);

            st = count_statistic(h_b, size);

            printf("\nStatistical parameters for matrix B\n");
            printf("Mean of matrix: %.2f\n", st.mean);
            printf("Standard deviation of matrix: %.2f\n", st.std);
            printf("Confidence interval with 95%% confidence: (%.2f, %.2f)\n", st.confi_inter_left, st.confi_inter_right);

        }
    }


}

int main(int argc, char const* argv[])
{
    algorith_main(1000);
    return 0;
}